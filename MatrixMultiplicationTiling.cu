
#include <hip/hip_runtime.h>
#include<algorithm>
#include<cassert>
#include<cstdlib>
#include<functional>
#include<iostream>
#include<vector>


using std::cout;
using std::generate;
using std::vector;

const int N = 1<<10;
const int SHMEM_SIZE = 1<<10;



__global__ void matrixMul(const int *a, const int *b, int *c) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Statically allocated shared memory
  __shared__ int s_a[SHMEM_SIZE];
  __shared__ int s_b[SHMEM_SIZE];

  // Accumulate in temporary variable
  int tmp = 0;

  // Sweep tile across matrix
  for (int i = 0; i < N; i += blockDim.x) {
    // Load in elements for this tile
    s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * N + i + threadIdx.x];
    s_b[threadIdx.y * blockDim.x + threadIdx.x] =
        b[i * N + threadIdx.y * N + col];

    // Wait for both tiles to be loaded in before doing computation
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp +=
          s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    }

    // Wait for all threads to finish using current tiles before loading in new
    // ones
    __syncthreads();
  }

  // Write back results
  c[row * N + col] = tmp;
}
 void verify_results(vector<int> &a, vector<int> &b, vector<int> &c) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[i * N + k] * b[k * N + j];
      }

      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
  }
}



int main(){
    // Matrix size of 1024x1024
    // int N = 1<<10;


    size_t bytes = N*N * sizeof(int);
    vector<int>h_a(N*N);
    vector<int>h_b(N*N);
    vector<int>h_c(N*N);

    generate(h_a.begin(), h_a.end(), [](){return rand()%100;});
    generate(h_b.begin(), h_b.end(), [](){return rand()%100;});


    int *d_a, *d_b, *d_c;

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);


    int Thread = 32;

    int BLOCKS = N/Thread;

    dim3 threads(Thread, Thread);
    dim3 blocks(BLOCKS, BLOCKS);

    matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);

    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    verify_results(h_a, h_b, h_c);



    for(int i = 0; i<4;i++){
        cout<< h_c[i]<<" ";
    }
    cout<<"\nCompleted Successflly\n";

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}