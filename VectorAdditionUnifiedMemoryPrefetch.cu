#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <iostream>
#include <vector>



__global__ void vectorAdd(const int* __restrict a, const int *__restrict b, int* __restrict c, int N) {
	int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

	if (tid < N)
		c[tid] = a[tid] + b[tid];
}

void allocateMemory(int *&a, int *&b, int *&c, int N){
    
    for(int i = 0; i<N; i++){
        assert(c[i]==a[i]+b[i]);
    }
}



int main() {
	constexpr int N = 1 << 16;
	constexpr size_t bytes = sizeof(int) * N;

	int *a, *b, *c;

    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    // Get the device ID for prefetching calls
    int id = hipGetDevice(&id);

    // Set some hints about the data and do some prefetching
    hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, id);




	for (int i = 0; i < N; i++) {
		a[i] = rand() % 100;
		b[i] = rand() % 100;
	}

    hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
    hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
    hipMemPrefetchAsync(a, bytes, id);
    hipMemPrefetchAsync(b, bytes, id);

	int NUM_THREAD = 1 << 10;
	int NUM_BLOCKS = (N + NUM_THREAD - 1) / NUM_THREAD;


	vectorAdd <<<NUM_BLOCKS, NUM_THREAD >>> (a, b, c, N);

    hipDeviceSynchronize();

    hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);
	
    allocateMemory(a, b, c, N);

    // for(int i = 0; i<N; i++){
    //     assert(c[i]==a[i]+b[i]);
    // }
	


    for (int i = 0; i < 10; i++)
		std::cout << c[i] << " ";
	hipFree(a);
	hipFree(b);
	hipFree(c);


	

	std::cout << "Completed Successfully";

	return 0;

}